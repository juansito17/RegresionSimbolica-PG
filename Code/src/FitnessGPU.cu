#include "hip/hip_runtime.h"
#include "FitnessGPU.cuh"
#include "Globals.h"
#include <hip/hip_runtime.h>
#include <math.h>

// Helper function to linearize the tree into a post-order array
void linearize_tree(const NodePtr& node, std::vector<LinearGpuNode>& linear_tree) {
    if (!node) {
        return;
    }
    linearize_tree(node->left, linear_tree);
    linearize_tree(node->right, linear_tree);
    linear_tree.push_back({node->type, node->value, node->op});
}

#if USE_GPU_ACCELERATION_DEFINED_BY_CMAKE
// CUDA kernel to evaluate a linearized tree
__global__ void calculate_raw_fitness_kernel(const LinearGpuNode* d_linear_tree,
                                             int tree_size,
                                             const double* d_targets,
                                             const double* d_x_values,
                                             size_t num_points,
                                             double* d_raw_fitness_results) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_points) {
        double x_val = d_x_values[idx];
        double stack[64]; // Max tree depth
        int stack_top = -1;

        for (int i = 0; i < tree_size; ++i) {
            LinearGpuNode node = d_linear_tree[i];
            if (node.type == NodeType::Constant) {
                stack[++stack_top] = node.value;
            } else if (node.type == NodeType::Variable) {
                stack[++stack_top] = x_val;
            } else if (node.type == NodeType::Operator) {
                double right = stack[stack_top--];
                double left = stack[stack_top--];
                double result;
                switch (node.op) {
                    case '+': result = left + right; break;
                    case '-': result = left - right; break;
                    case '*': result = left * right; break;
                    case '/':
                        if (fabs(right) < 1e-9) { // Avoid division by zero
                            result = HUGE_VAL;
                        } else {
                            result = left / right;
                        }
                        break;
                    default: result = NAN; break;
                }
                stack[++stack_top] = result;
            }
        }

        double predicted_val = (stack_top == 0) ? stack[0] : NAN;

        if (isnan(predicted_val) || isinf(predicted_val)) {
            d_raw_fitness_results[idx] = HUGE_VAL; // Assign a large error for invalid results
        } else {
            double diff = predicted_val - d_targets[idx];
            d_raw_fitness_results[idx] = diff * diff;
        }
    }
}

// CUDA kernel for parallel reduction (summation)
__global__ void reduce_sum_kernel(double* d_data, int N) {
    extern __shared__ double sdata[]; // Shared memory for reduction

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[tid] = (i < N) ? d_data[i] : 0.0; // Load data into shared memory

    __syncthreads(); // Synchronize threads in block

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) { // Write result back to global memory (first element of block)
        d_data[blockIdx.x] = sdata[0];
    }
}

// Host-side wrapper function to launch the CUDA kernel
double evaluate_fitness_gpu(NodePtr tree,
                            const std::vector<double>& targets,
                            const std::vector<double>& x_values,
                            double* d_targets, double* d_x_values) {
    if (x_values.size() != targets.size() || x_values.empty()) return INF;

    // Linearize the tree
    std::vector<LinearGpuNode> h_linear_tree;
    linearize_tree(tree, h_linear_tree);
    int tree_size = h_linear_tree.size();

    if (tree_size == 0) {
        return INF;
    }

    size_t num_points = x_values.size();
    LinearGpuNode* d_linear_tree;
    double* d_raw_fitness_results; // This will hold individual errors and then the final sum

    hipMalloc((void**)&d_linear_tree, tree_size * sizeof(LinearGpuNode));
    hipMalloc((void**)&d_raw_fitness_results, num_points * sizeof(double));

    hipMemcpy(d_linear_tree, h_linear_tree.data(), tree_size * sizeof(LinearGpuNode), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (num_points + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel to calculate individual squared errors
    calculate_raw_fitness_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_linear_tree, tree_size, d_targets, d_x_values, num_points, d_raw_fitness_results
    );
    hipDeviceSynchronize(); // Ensure kernel completes before reduction

    // --- Perform reduction on the GPU ---
    int current_size = num_points;
    while (current_size > 1) {
        int next_blocks_per_grid = (current_size + threadsPerBlock - 1) / threadsPerBlock;
        // Use shared memory for reduction, size is threadsPerBlock * sizeof(double)
        reduce_sum_kernel<<<next_blocks_per_grid, threadsPerBlock, threadsPerBlock * sizeof(double)>>>(
            d_raw_fitness_results, current_size
        );
        hipDeviceSynchronize(); // Ensure reduction step completes
        current_size = next_blocks_per_grid; // The result is in the first `next_blocks_per_grid` elements
    }

    double sum_sq_error_gpu = 0.0;
    hipMemcpy(&sum_sq_error_gpu, d_raw_fitness_results, sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_linear_tree);
    hipFree(d_raw_fitness_results);

    // Check for invalid results (propagated from kernel)
    if (isinf(sum_sq_error_gpu) || isnan(sum_sq_error_gpu)) {
        return INF;
    }

    double raw_fitness;
    if (USE_RMSE_FITNESS) {
        if (num_points == 0) return INF;
        double mse = sum_sq_error_gpu / num_points;
        raw_fitness = sqrt(mse);
    } else {
        raw_fitness = sum_sq_error_gpu;
    }

    double complexity = static_cast<double>(::tree_size(tree));
    double penalty = complexity * COMPLEXITY_PENALTY_FACTOR;
    double final_fitness = raw_fitness * (1.0 + penalty);

    if (isnan(final_fitness) || isinf(final_fitness) || final_fitness < 0) {
        return INF;
    }

    return final_fitness;
}
#endif // USE_GPU_ACCELERATION_DEFINED_BY_CMAKE
